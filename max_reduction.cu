#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/*
    block_max_reduction_kernel 

    Kernel to find the max across all threads in a block 

    Inputs: matrix of values and matrix to hold max value(s) for each block
    *** MUST BE PADDED TO MULTIPLE OF 32 IN DIMENSION BEING REDUCED
    *** CURRENTLY ONLY FINDS MAX OF POSITIVE NUMBERS
    
    Outputs: None...new matrix with one less dimension filled with the max of each block 

*/
__global__ void block_max_reduction_kernel(double* grid, double* row_max_grid) {

    // shared memory so we can compare the max across warps
    // residual matrix was padded to have dimensions in multiples of 32 so
    // length of array is number of threads / 32 i.e. we can store a max for each
    // up to 32 warps (since max threads is 1024)
    extern __shared__ double warp_max[];

    // residual at grid point and variable to hold value to compare to 
    double x = grid[blockIdx.x * blockDim.x + threadIdx.x];
    double x_comp;
    
    // do comparisons in chunks of 32
    // each set of 32 points will have max across those 32 points
    int mask, level;
    for (mask = 1, level = 0; level < 6; mask *= 2, ++level) {
        x_comp = __shfl_xor_sync(0xFFFFFFFF, x, mask);
        x = x > x_comp ? x : x_comp;
    }

    // fill shared memory array with maxes of each 32 element warp
    if (threadIdx.x % 32 == 0) {
        warp_max[threadIdx.x / warpSize] = x;
    }

    __syncthreads();

    // get max across warps in a given block 
    // first threads from 0 to (num threads / 32) - 1 get values stored in warp_max and
    // every other thread stores 0 
    x = (threadIdx.x < blockDim.x / warpSize) ? warp_max[threadIdx.x] : 0;

    // find the max for this block 
    for (mask = 1, level = 0; level < 6; mask *= 2, ++level) {
        x_comp = __shfl_xor_sync(0xFFFFFFFF, x, mask);
        x = x > x_comp ? x : x_comp;
    }

    // save max of block to output array
    if (threadIdx.x == 0) {
        row_max_grid[blockIdx.x] = x;
    } 
}

int main(int argc, char* argv[]) {

    // device handle and properties
    int dev;
    hipDeviceProp_t prop;

    // set all entries to 0 to mean no preference
    memset(&prop, 0, sizeof(hipDeviceProp_t));

    // look for GPU with at least 13 cores
    prop.multiProcessorCount = 13;

    // choose and set GPU
    hipChooseDevice(&dev, &prop);
    hipSetDevice(dev);

    // x and y dimensions
    int N_x = 34;
    int N_y = 34; 

    // pad to get multiples of 32
    N_x = N_x % 32 == 0 ? N_x : N_x + 32 - (N_x % 32); 
    N_y = N_y % 32 == 0 ? N_y : N_y + 32 - (N_y % 32); 

    // create grid grid to take max over 
    double* grid = (double *) malloc(N_x * N_y * sizeof(double));

    for (int i = 0; i < N_y; ++i) {
        for (int j = 0; j < N_x; ++j) {
            grid[i * N_x + j]  = i * N_x + j;        
        }
    }

    // copy array to device
    double* dev_grid;
    hipMalloc((void**)&dev_grid, N_x * N_y * sizeof(double));
    hipMemcpy(dev_grid, grid, N_x * N_y * sizeof(double), hipMemcpyHostToDevice);

    // allocate memory for grid at each point and for max grid for each row on device
    double* dev_row_max_grid; 
    hipMalloc((void**)&dev_row_max_grid, N_y * sizeof(double));

    // max grid on host and device 
    double max_grid; 
    double* dev_max_grid;
    hipMalloc((void**)&dev_max_grid, sizeof(double));

    // call kernel
    block_reduction<<<N_y, N_x, 32 * sizeof(double)>>>(dev_grid, dev_row_max_grid);
    block_reduction<<<1, N_y, 32 * sizeof(double)>>>(dev_row_max_grid, dev_max_grid);

    // copy array of row maxes and overall max back to host
    double* row_max_grid = (double *) malloc(N_y * sizeof(double));
    hipMemcpy(row_max_grid, dev_row_max_grid, N_y * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&max_grid, dev_max_grid, sizeof(double), hipMemcpyDeviceToHost);

    printf("Max: %f\n", max_grid);

    // free memory on device 
    hipFree(dev_grid);
    hipFree(dev_max_grid);
    hipFree(dev_row_max_grid);

    // free memory on host 
    free(grid);

    return 0;

}
